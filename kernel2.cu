#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "matrix.h"

#define TILE_SIZE 32
#define MAX_ELEMS_PER_COL 16
#define MAX_NNZ_OUTPUT 10000000u
#define COARSEN 2   // rows per thread

__device__ static void clear_marker_row(int *m, unsigned int colsB) {
    for (unsigned int j = 0; j < colsB; ++j) {
        m[j] = -1;
    }
}

// Coarsened SpMSpM kernel
__global__ static void spmspm_gpu2_kernel(
    unsigned int rowsA,
    const unsigned int *A_rowPtrs,
    const unsigned int *A_colIdxs,
    const float *A_vals,
    const unsigned int *B_rowPtrs,
    const unsigned int *B_colIdxs,
    const float *B_vals,
    unsigned int colsB,
    unsigned int *C_rowIdxs,
    unsigned int *C_colIdxs,
    float *C_vals,
    int *marker,
    unsigned int *d_outputcount
) {
    unsigned int tid      = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int startRow = tid * COARSEN;

    __shared__ unsigned int s_row_ind[TILE_SIZE][MAX_ELEMS_PER_COL];
    __shared__ float s_vals [TILE_SIZE][MAX_ELEMS_PER_COL];

    // Process up to COARSEN rows
    for (int r = 0; r < COARSEN; ++r) {
        unsigned int i = startRow + r;
        if (i >= rowsA) break;

        // marker for this row
        int *m = marker + i * colsB;
        clear_marker_row(m, colsB);

        // multiply by B and accumulate
        unsigned int rowStart = A_rowPtrs[i];
        unsigned int rowEnd   = A_rowPtrs[i + 1];
        for (unsigned int pa = rowStart; pa < rowEnd; ++pa) {
            float vA = A_vals[pa];
            unsigned int k = A_colIdxs[pa];

            unsigned int b0 = B_rowPtrs[k];
            unsigned int b1 = B_rowPtrs[k + 1];
            unsigned int len = b1 - b0;
            unsigned int limit = (len < MAX_ELEMS_PER_COL ? len : MAX_ELEMS_PER_COL);

            // load full row up to limit for this thread
            for (unsigned int pb = 0; pb < limit; ++pb) {
                s_row_ind[threadIdx.x][pb] = B_colIdxs[b0 + pb];
                s_vals[threadIdx.x][pb] = B_vals[b0 + pb];
            }
            __syncthreads();

            // perform dot-product-like accumulation
            for (unsigned int pb = 0; pb < limit; ++pb) {
                unsigned int j    = s_row_ind[threadIdx.x][pb];
                float        prod = vA * s_vals    [threadIdx.x][pb];
                int          mi   = m[j];
                if (mi == -1) {
                    unsigned int idx = atomicAdd(d_outputcount, 1u);
                    if (idx < MAX_NNZ_OUTPUT) {
                        C_rowIdxs[idx] = i;
                        C_colIdxs[idx] = j;
                        C_vals   [idx] = prod;
                        m[j] = idx;
                    }
                } else {
                    C_vals[mi] += prod;
                }
            }
            __syncthreads();
        }
    }
}

void spmspm_gpu2(
    COOMatrix* cooMatrix1,
    CSRMatrix* csrMatrix1,
    CSCMatrix* cscMatrix1,
    COOMatrix* cooMatrix2,
    CSRMatrix* csrMatrix2,
    CSCMatrix* cscMatrix2,
    COOMatrix* cooMatrix3,
    unsigned int numRows1,
    unsigned int numRows2,
    unsigned int numCols2,
    unsigned int numNonzeros1,
    unsigned int numNonzeros2
) {
    CSRMatrix hA; CUDA_ERROR_CHECK(hipMemcpy(&hA, csrMatrix1, sizeof(hA), hipMemcpyDeviceToHost));
    const unsigned int *d_A_rowPtrs = hA.rowPtrs;
    const unsigned int *d_A_colIdxs = hA.colIdxs;
    const float        *d_A_vals    = hA.values;

    CSRMatrix hB; CUDA_ERROR_CHECK(hipMemcpy(&hB, csrMatrix2, sizeof(hB), hipMemcpyDeviceToHost));
    const unsigned int *d_B_rowPtrs = hB.rowPtrs;
    const unsigned int *d_B_colIdxs = hB.colIdxs;
    const float        *d_B_vals    = hB.values;

    COOMatrix hC; CUDA_ERROR_CHECK(hipMemcpy(&hC, cooMatrix3, sizeof(hC), hipMemcpyDeviceToHost));
    unsigned int *d_C_rowIdxs = hC.rowIdxs;
    unsigned int *d_C_colIdxs = hC.colIdxs;
    float        *d_C_vals    = hC.values;

    int          *d_marker;
    unsigned int *d_outputcountt;
    CUDA_ERROR_CHECK(hipMalloc(&d_marker, numRows1 * numCols2 * sizeof(int)));
    CUDA_ERROR_CHECK(hipMalloc(&d_outputcountt,    sizeof(unsigned int)));
    CUDA_ERROR_CHECK(hipMemset(d_outputcountt, 0,  sizeof(unsigned int)));

    // Launch coarsened kernel
    int threads = TILE_SIZE;
    int blocks  = (numRows1 + threads * COARSEN - 1) / (threads * COARSEN);
    spmspm_gpu2_kernel<<<blocks, threads>>>(
        numRows1,
        d_A_rowPtrs, d_A_colIdxs, d_A_vals,
        d_B_rowPtrs, d_B_colIdxs, d_B_vals,
        numCols2,
        d_C_rowIdxs, d_C_colIdxs, d_C_vals,
        d_marker,
        d_outputcountt
    );
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    unsigned int newCount;
    CUDA_ERROR_CHECK(hipMemcpy(&newCount, d_outputcountt, sizeof(newCount), hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK(hipMemcpy(
        (char*)cooMatrix3 + offsetof(COOMatrix, numNonzeros),
        &newCount,
        sizeof(newCount),
        hipMemcpyHostToDevice
    ));

    CUDA_ERROR_CHECK(hipFree(d_marker));
    CUDA_ERROR_CHECK(hipFree(d_outputcountt));
}
